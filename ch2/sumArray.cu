
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <stdio.h>


__global__ void sumArrayOnDevice(float *A, float *B, float *C, int n)
{
	for (int i = 0; i < n; i++)
	{
		C[i] = A[i] + B[i];
	}
}

void sumArrayOnHost(float *A, float *B, float *C, int n)
{
	for (int i = 0; i < n; i++)
	{
		C[i] = A[i] + B[i];
	}
}

void initData(float *ptr, int size)
{
	time_t t;
	srand((unsigned int)time(&t));
	for (int i = 0; i < size; i++)
	{
		ptr[i] = (float)(rand() & 0xFF) / 10.0f;
	}
}
void printArray(float* ptr, int size)
{
	for (int i = 0; i < size; i++)
		printf("%f ", ptr[i]);
}
int main()
{
	int nElem = 1024;
	size_t nBytes = nElem * sizeof(float);
	float *A, *B, *C, *D;
	A = (float*)malloc(nBytes);
	B = (float*)malloc(nBytes);
	C = (float*)malloc(nBytes);
	D = (float*)malloc(nBytes);

	float *cA, *cB, *cC;
	hipMalloc((float**)&cA, nBytes);
	hipMalloc((float**)&cB, nBytes);
	hipMalloc((float**)&cC, nBytes);

	initData(A, nElem);
	initData(B, nElem);

	hipMemcpy(cA, A, nBytes, hipMemcpyHostToDevice);
	hipMemcpy(cB, A, nBytes, hipMemcpyHostToDevice);
	
	sumArrayOnDevice<<<1,1>>>(cA, cB, cC, nElem);

	//	get all 0 data, just init data
	//printArray(C, nElem);
	//printf("\n\n\n\n\n");

	//	block, wait for answer
	hipMemcpy(C, cC, nBytes, hipMemcpyDeviceToHost);
	printArray(C, nElem);

	//	do same in cpu, check answer
	printf("\n\n\n\n\n");
	sumArrayOnHost(A,B,D,nElem);
	printArray(D, nElem);

	free(A);
	free(B);
	free(C);
	free(D);

	hipFree(cA);
	hipFree(cB);
	hipFree(cC);

	return 0;
}