
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <stdio.h>


__global__ void sumArrayOnDevice(float *A, float *B, float *C, int n)
{
	for (int i = 0; i < n; i++)
	{
		C[i] = A[i] + B[i];
	}
}

void sumArrayOnHost(float *A, float *B, float *C, int n)
{
	for (int i = 0; i < n; i++)
	{
		C[i] = A[i] + B[i];
	}
}

void initData(float *ptr, int size)
{
	time_t t;
	srand((unsigned int)time(&t));
	for (int i = 0; i < size; i++)
	{
		ptr[i] = (float)(rand() & 0xFF) / 10.0f;
	}
}
void printArray(float* ptr, int size)
{
	for (int i = 0; i < size; i++)
		printf("%f ", ptr[i]);
}
int main()
{
	int nElem = 1024;
	size_t nBytes = nElem * sizeof(float);
	float *A, *B, *C;
	A = (float*)malloc(nBytes);
	B = (float*)malloc(nBytes);
	C = (float*)malloc(nBytes);

	float *cA, *cB, *cC;
	hipMalloc((float**)&cA, nBytes);
	hipMalloc((float**)&cB, nBytes);
	hipMalloc((float**)&cC, nBytes);

	initData(A, nElem);
	initData(B, nElem);

	hipMemcpy(A, cA, nBytes, hipMemcpyHostToDevice);
	hipMemcpy(B, cB, nBytes, hipMemcpyHostToDevice);

	//sumArrayOnHost(A,B,C,nElem);
	
	sumArrayOnDevice<<<1,10>>>(A, B, C, nElem);
	hipMemcpy(cC, C, nBytes, hipMemcpyDeviceToHost);
	
	// maybe gpu not finish,TODO!!!
	printArray(C, nElem);
	free(A);
	free(B);
	free(C);

	hipFree(cA);
	hipFree(cB);
	hipFree(cC);

	return 0;
}